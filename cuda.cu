
#include <hip/hip_runtime.h>
#include <atomic>
#include <chrono>
#include <stddef.h>
#include <stdio.h>
#include <stdint.h>
#include <sys/time.h>
#include <pthread.h>
#include <thread>

#define GPUS 2
#define BYTE unsigned char
#define THREADS 64
#define BLOCKS 256

inline BYTE *create_bitarray(size_t elements) {
    size_t bytes = elements / 8 + ((elements % 8) > 1);
    return (BYTE *) calloc(bytes, 1);
}

inline BYTE bitarray_get(BYTE *bitarray, size_t index) {
    size_t byte = index / 8;
    size_t offset_mask = 1 << (7 - (index % 8));
    return bitarray[byte] & offset_mask;
}

inline void bitarray_set(BYTE *bitarray, size_t index) {
    size_t byte = index / 8;
    size_t offset_mask = 1 << (7 - (index % 8));
    bitarray[byte] |= offset_mask;
}

void naive_sieve(uint64_t upper_bound, uint64_t **primes, uint64_t *prime_count) {
    BYTE *is_prime = create_bitarray(upper_bound);
    uint64_t total_primes = upper_bound - 2;
    for (uint64_t p = 2; (p * p) < upper_bound; p++) {
        if (bitarray_get(is_prime, p) == 0) {
            for (uint64_t i = p * p; i < upper_bound; i += p) {
                if (bitarray_get(is_prime, i) == 0) {
                    total_primes--;
                    bitarray_set(is_prime, i);
                }
            }
        }
    }

    *prime_count = total_primes;
    *primes = (uint64_t *) malloc(sizeof(uint64_t) * total_primes);
    size_t i = 0;
    for (uint64_t p = 2; p < upper_bound; p++) {
        if (bitarray_get(is_prime, p) == 0) (*primes)[i++] = p;
    }

    free(is_prime);
}

inline BYTE *create_bitarrays_gpu(size_t elements, size_t *bytes_per_bitarray, size_t bitarrays) {
    *bytes_per_bitarray = elements / 8 + ((elements % 8) > 1);
    BYTE *bitarrays_mem;
    hipMalloc(&bitarrays_mem, *bytes_per_bitarray * bitarrays);
    return bitarrays_mem;
}

__global__ void sieve_chunk_gpu(BYTE *is_prime_arrays, size_t is_prime_bytes,
                                uint64_t default_prime_count, uint64_t *prime_counts,
                                uint64_t *seed_primes, uint64_t seed_count,
                                uint64_t chunk_size, uint64_t chunk_count, uint64_t chunk_offset) {
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= chunk_count) return;
    uint64_t offset_index = index + chunk_offset;

    BYTE *is_prime = is_prime_arrays + index * is_prime_bytes;
    
    uint64_t low = (offset_index + 1) * chunk_size;
    uint64_t high = low + chunk_size;
    low += 1 - (low & 1);
    high -= 1 - (high & 1);

    for (size_t i = 0; i < is_prime_bytes; i++) is_prime[i] = 0;

    uint64_t prime_count = default_prime_count;
    for (size_t i = 1; i < seed_count; i++) {
        uint64_t low_multiple = (uint64_t) (floor((double) low / (double) seed_primes[i]) * (double) seed_primes[i]);
        low_multiple += (low_multiple < low) * seed_primes[i];

        uint64_t j = low_multiple + (1 - (low_multiple & 1)) * seed_primes[i];
        while (j <= high) {
            uint64_t j_idx = (j - low) / 2;
            size_t byte = j_idx / 8;
            size_t offset_mask = 1 << (7 - (j_idx % 8));
            uint64_t is_set = (is_prime[byte] & offset_mask) == 0;
            prime_count -= is_set;
            BYTE potential_new_value = is_prime[byte] | offset_mask;
            is_prime[byte] = is_set * potential_new_value + (1 - is_set) * is_prime[byte];
            j += seed_primes[i] * 2;
        }
    }

    prime_counts[index] = prime_count;
}

uint64_t sieve_chunk_cpu(BYTE *is_prime, uint64_t *seed_primes, uint64_t seed_count, uint64_t chunk_size, uint64_t chunk_index) {
    uint64_t low = (chunk_index + 1) * chunk_size;
    uint64_t high = low + chunk_size;
    low += 1 - (low & 1);
    high -= 1 - (high & 1);

    for (size_t i = 1; i < seed_count; i++) {
        uint64_t low_multiple = (uint64_t) (floor((double) low / (double) seed_primes[i]) * (double) seed_primes[i]);
        low_multiple += (low_multiple < low) * seed_primes[i];

        uint64_t j = low_multiple + (1 - (low_multiple & 1)) * seed_primes[i];
        while (j <= high) {
            uint64_t j_idx = (j - low) / 2;
            size_t byte = j_idx / 8;
            size_t offset_mask = 1 << (7 - (j_idx % 8));
            uint64_t is_set = (is_prime[byte] & offset_mask) == 0;
            BYTE potential_new_value = is_prime[byte] | offset_mask;
            is_prime[byte] = is_set * potential_new_value + (1 - is_set) * is_prime[byte];
            j += seed_primes[i] * 2;
        }
    }

    return low;
}

typedef struct {
    int gpu;

    uint64_t chunk_size;
    uint64_t chunk_count;
    uint64_t chunk_offset;
    uint64_t chunk_prime_count;

    uint64_t *seed_primes;
    uint64_t seed_prime_count;

    uint64_t *chunk_prime_counts;

    std::atomic<uint64_t> *processed_chunks;
} gpu_worker_input;

void *process_chunks_on_gpu(void *vinput) {
    gpu_worker_input *input = (gpu_worker_input *) vinput;

    hipSetDevice(input->gpu);

    uint64_t kernel_chunk_count = THREADS * BLOCKS;
    uint64_t invocations = 1;
    if (input->chunk_count < kernel_chunk_count) kernel_chunk_count = input->chunk_count;
    else invocations = input->chunk_count / kernel_chunk_count + ((input->chunk_count % kernel_chunk_count) > 0);

    uint64_t *seed_primes_gpu;
    size_t seed_primes_size = sizeof(uint64_t) * input->seed_prime_count;
    hipMalloc(&seed_primes_gpu, seed_primes_size);
    hipMemcpy(seed_primes_gpu, input->seed_primes, seed_primes_size, hipMemcpyHostToDevice);

    size_t is_prime_bytes;
    BYTE *is_prime_arrays_gpu = create_bitarrays_gpu(input->chunk_prime_count, &is_prime_bytes, kernel_chunk_count);

    uint64_t *prime_counts_gpu;
    hipMalloc(&prime_counts_gpu, sizeof(uint64_t) * kernel_chunk_count);

    uint64_t total_chunks_processed = 0;
    for (uint64_t i = 0; i < invocations; i++) {
        uint64_t offset = i * kernel_chunk_count;
        uint64_t remaining_chunks = input->chunk_count - offset;
        if (remaining_chunks > kernel_chunk_count) remaining_chunks = kernel_chunk_count;
        offset += input->chunk_offset;

        sieve_chunk_gpu<<<THREADS, BLOCKS>>>(
            is_prime_arrays_gpu, is_prime_bytes,
            input->chunk_prime_count, prime_counts_gpu,
            seed_primes_gpu, input->seed_prime_count,
            input->chunk_size, remaining_chunks, offset
        );
        hipDeviceSynchronize();

        hipMemcpy(input->chunk_prime_counts + total_chunks_processed, prime_counts_gpu, sizeof(uint64_t) * remaining_chunks, hipMemcpyDeviceToHost);
        total_chunks_processed += remaining_chunks;
        *input->processed_chunks += remaining_chunks;
    }

    hipFree(is_prime_arrays_gpu);
    hipFree(seed_primes_gpu);

    return NULL;
}

uint64_t sieve(uint64_t m) {
    double dm = (double) m;
    uint64_t upper_bound = (uint64_t) (dm * log(dm)) + (dm * log(log(dm)));
    uint64_t chunk_size = (uint64_t) sqrt((double) upper_bound);
    uint64_t chunk_count = chunk_size - 1;
    uint64_t chunk_prime_count = chunk_size / 2 + chunk_size % 2;

    printf("Must process %lu chunks. Checking up to %lu.\n", chunk_count, upper_bound);

    uint64_t *seed_primes;
    uint64_t seed_prime_count;
    naive_sieve(chunk_size, &seed_primes, &seed_prime_count);

    uint64_t chunks_per_gpu = chunk_count / GPUS;
    uint64_t last_gpu_overflow = chunk_count % GPUS;
    uint64_t *chunk_prime_counts[GPUS];
    gpu_worker_input inputs[GPUS];
    pthread_t tids[GPUS];
    std::atomic<uint64_t> total_chunks_processed(0);

    for (size_t i = 0; i < GPUS; i++) {
        uint64_t gpu_chunk_count = chunks_per_gpu;
        if (i == (GPUS - 1)) gpu_chunk_count += last_gpu_overflow;

        chunk_prime_counts[i] = (uint64_t *) calloc(sizeof(uint64_t), gpu_chunk_count);

        inputs[i] = {
            (int) i,

            chunk_size,
            gpu_chunk_count,
            chunks_per_gpu * i,
            chunk_prime_count,

            seed_primes,
            seed_prime_count,

            chunk_prime_counts[i],

            &total_chunks_processed
        };

        pthread_create(tids + i, NULL, process_chunks_on_gpu, inputs + i);
    }

    while (total_chunks_processed < chunk_count) {
        printf("Processed: %lu\r", total_chunks_processed.load());
        std::this_thread::sleep_for(std::chrono::milliseconds(100));
    }
    printf("\n");

    for (size_t i = 0; i < GPUS; i++) pthread_join(tids[i], NULL);

    uint64_t total_chunks_checked = 0;
    uint64_t found_primes = seed_prime_count;
    for (size_t i = 0; i < GPUS; i++) {
        uint64_t gpu_chunk_count = chunks_per_gpu;
        if (i == (GPUS - 1)) gpu_chunk_count += last_gpu_overflow;

        for (size_t j = 0; j < gpu_chunk_count; j++) {
            uint64_t new_found = found_primes + chunk_prime_counts[i][j];
            if (new_found >= m) goto found_chunk;
            found_primes = new_found;
            total_chunks_checked++;
        }
    }

    BYTE *is_prime;
    uint64_t chunk_low;

    printf("Couldn't find nth prime.\n");
    goto cleanup;

found_chunk:
    is_prime = create_bitarray(chunk_prime_count);
    chunk_low = sieve_chunk_cpu(is_prime, seed_primes, seed_prime_count, chunk_size, total_chunks_checked);

    for (size_t i = 0; i < chunk_prime_count; i++) {
        if (bitarray_get(is_prime, i) == 0) found_primes++;
        if (found_primes == m) {
            printf("%lu\n", chunk_low + i * 2);
            break;
        }
    }

    free(is_prime);

cleanup:
    free(seed_primes);
    return chunks_per_gpu;
}

int main() {
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);
    sieve(1e9);
    gettimeofday(&tv2, NULL);
    printf("Total time = %f seconds\n", (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec));
    return sieve(1e8);
} 
